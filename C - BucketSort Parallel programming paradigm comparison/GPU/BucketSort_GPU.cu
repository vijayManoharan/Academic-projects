#include "hip/hip_runtime.h"
///* 
// * File:   BucketSort.c
// * Author: vijay manoharan
// *. C program for running bucket sort on CUDA.
// * Created on November 8, 2014, 8:19 PM
// */

#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include <stdint.h>

//header file for random value generation.
#include "rnd.h"
#include "rnd.c"

//headed file for cuda
#include <hip/hip_runtime.h>
int size;

/*
 * Using quickselect to partially sort the array
 * algorithm refered from wikipedia "http://en.wikipedia.org/wiki/Quickselect"
 * Implementation is done by me. 
 */

int partition(float *a, int left, int right, int pivot) {
    float pivotValue = a[pivot], temp;
    int index = left;
    int i;

    //move pivot to rightmost of the array
    a[pivot] = a[right];
    a[right] = pivotValue;

    /* We start from left of the array and keep comparing the values.
     *  If we find any value greater than pivot we swap the pivot with that value.
     *  this is done so that the value on the left are always lesser than pivot
     */
    for (i = left; i < right; i++) {
        if (a[i] < pivotValue) {
            temp = a[i];
            a[i] = a[index];
            a[index] = temp;
            index++;
        }
    }
    /*move pivot back to its initial position
     */
    temp = a[index];
    a[index] = a[right];
    a[right] = temp;
    return index;

}

/*the main aspect of the function is 
 * find the pivot value which is pivot given and return the value
 * next thing is all the value in the left index of pivot index is smaller 
 * all the values in the right index is greater.
 */
float quickSelect(float *a, int left, int right, int pivot) {

    //base condition if list contains one element return that
    if (left == right)
        return a[left];

    /* select the pivot index between left and right
     * i am selecting the mid value
     */

    int pivotindex = (right + left) / 2;
    pivotindex = partition(a, left, right, pivotindex);
    //int size_leftarray = pivotposition - low + 1;
    //pivot in its correct position
    if (pivot == pivotindex)
        return a[pivotindex];
    else if (pivot < pivotindex)
        return quickSelect(a, left, pivotindex - 1, pivot);
    else
        return quickSelect(a, pivotindex + 1, right, pivot);
}

//merging the sorted left and right of the merge sort

__device__ void merge(float *a, int low, int mid, int high) {
    //printf("inside merge\n");
    int i = 0;
    int left = low, right = mid + 1;
    int j;
	float *temp =(float*) malloc((high - low + 1 )* sizeof(*temp) );
    
    /* Create a temp sorted list
     * get the min of left part and right part
     * if min is in the left part iterate left or else iterate right
     */

    while ((left <= mid)&&(right <= high)) {
        if (a[left] < a[right])
            temp[i++] = a[left++];
        else
            temp[i++] = a[right++];
    }
    //left part contains larger values
    while (left <= mid)
        temp[i++] = a[left++];
    //right part contains larger values
    while (right <= high)
        temp[i++] = a[right++];


    //copy the sorted values
    for (j = 0; j < i; j++) {
        a[low + j] = temp[j];
    }
}
//merge sort algorithm
void Quick(float *a, int low, int mid, int high) {
    //printf("inside merge\n");
    int i = 0;
    int left = low, right = mid + 1;
    int j;
	float *temp =(float*) malloc((high - low + 1 )* sizeof(*temp) );
    
    /* Create a temp sorted list
     * get the min of left part and right part
     * if min is in the left part iterate left or else iterate right
     */

    while ((left <= mid)&&(right <= high)) {
        if (a[left] < a[right])
            temp[i++] = a[left++];
        else
            temp[i++] = a[right++];
    }
    //left part contains larger values
    while (left <= mid)
        temp[i++] = a[left++];
    //right part contains larger values
    while (right <= high)
        temp[i++] = a[right++];


    //copy the sorted values
    for (j = 0; j < i; j++) {
        a[low + j] = temp[j];
    }
}

__device__ void Mergesort(float *a, int low, int high) {
    int mid;
    if (low < high) {
        mid = (low + high) / 2;
        Mergesort(a, low, mid);
        Mergesort(a, mid + 1, high);
        merge(a, low, mid, high);
    }
}

void Quickselect(float *a, int low, int high) {
    int mid;
    if (low < high) {
        mid = (low + high) / 2;
        Quickselect(a, low, mid);
        Quickselect(a, mid + 1, high);
        Quick(a, low, mid, high);
    }
}

int isnumber(int size) {
    int flag = 0;

    return flag;
}



__global__ void cudaBucketSort(float *array, int *pivots) {
    /* 1.call merge sort for each threads
     * 2.Range is given based on the pivot values.
     * 3.Each thread will sort its own data set.
     */
	if(threadIdx.x==0){ 
    int start = pivots[blockIdx.x],i;
    int end = pivots[blockIdx.x +1];
    printf("pivot index %d\t start %d \t end %d\n", (blockIdx.x ) , start, end);
    Mergesort(array, start, end);
	for(i=start;i<end;i++)
		 printf("%d-->> %f\n", i, array[i]);
	}
	
}

int main(int argc, char** argv) {

    if (strcmp("-t", argv[1]) != 0) {
        printf("invalid command line agrument\n");
        return 0;
    }


    size = atoi(argv[2]);
    
    //variables for size and number of threads, blocks etc.
    int BlockNum=10;
  
    int Range=size/10;
    
    int ThreadSize ;
    
	if(size<1000)
		ThreadSize=size;
	else
		ThreadSize=1000;
	
    //variables for pivots 
    int pivots[11];
    float pivots_value[11];
    struct timeval tv1, tv2;
    float array[size];
    printf("the size is %d", size);
    int i;
    
    //cuda device variables 

    float *dev_array;
	int	*dev_pivots;

  
    hipMalloc((void **) &dev_array, size * sizeof (float));
    hipMalloc((void **) &dev_pivots, (11) * sizeof (int));

    random_number_generator_normal(array, size, 10*size);
    //    for (i = 0; i < size; i++) {
    //        printf("%d-->> %f\t", i, array[i]);
    //    }
    //find 25th element

    gettimeofday(&tv1, NULL);

    for (i = 0; i < 10; i++) {
        pivots[i] = Range*i;
    }
    pivots[i] = size;
    pivots_value[0] = 0;
    pivots_value[i] = array[size - 1];
    for (i = 1; i < 10; i++) {
        pivots_value[i] = quickSelect(array, pivots[i - 1], size - 1, (pivots[i]));
		if(size>10000)
			 Quickselect(array, pivots[i-1], pivots[i]);
	   //printf("is the %d largest value %f\n", pivots[i], pivots_value[i]);
        //find 5th largest element

    }

    /* MergeSort here.
     * 
     */
    
    hipMemcpy(dev_array, array, size * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(dev_pivots, pivots, (11) * sizeof (int), hipMemcpyHostToDevice);
    cudaBucketSort <<<BlockNum, ThreadSize>>>(dev_array, dev_pivots);
	if(size<=10000)
    hipMemcpy(array, dev_array, size * sizeof (float), hipMemcpyDeviceToHost);

	//cuda function here.
	
    printf("\nmergesort starts \n");




    gettimeofday(&tv2, NULL);


    //sort(array);
    printf("sorted list is \n");
    for (i = 0; i < size; i++) {
       printf("%d-->> %f\n", i, array[i]);
    }
	
    printf("\nTotal time = %f seconds\n",
            (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
            (double) (tv2.tv_sec - tv1.tv_sec));
    return 0;


}