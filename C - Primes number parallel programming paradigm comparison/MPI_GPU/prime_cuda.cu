//program by Vijay manoharan SUNYB


//header files


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<math.h>
#include <stdbool.h>

#include<sys/time.h>
//end of header file


//cuda global function

__global__ void cudeprime(int limit,int *threadcounter, int *maxi,int pid,int Num_processor) {
    int  number, prime_div;
	number = 29+(blockIdx.x*2*blockDim.x)+(2*threadIdx.x*Num_processor)+(2*pid);
	//printf("number %d and process %d\n",number,pid);
	//printf("block -%d\t size -%d\tthread- %d\t the number is %d\n",blockIdx.x,blockDim.x,threadIdx.x,number);
	if(number>limit) return;
    if ((number % 3 == 0) || (number % 5 == 0) || (number % 7 == 0) || (number % 13 == 0) || (number % 11 == 0) || (number % 17 == 0) || (number % 19 == 0) || (number % 23 == 0)) {
    //do nothing 
    } else {
	prime_div = 29;
		while (prime_div * prime_div <= number) {

			if ((number % prime_div == 0))
				break;

			prime_div = prime_div + 2;
			
		}
		//if the number is not prime_divisible until the square root of the number then the number is prime.
		if (prime_div * prime_div > number) {
			//write the value to index.
			threadcounter[(blockIdx.x*blockDim.x)+threadIdx.x]=number;
			//printf("thread n number %d %d\n",threadIdx.x,number);
		}
	}
}


//main function

extern "C" int *cuda_main(int p_id, int Num_Processor,int number_of_prime) {
    printf("in cuda n %d pid %d  number of process %d \n",number_of_prime,p_id,Num_Processor);
	static int returnval[2];
    //variables for MPI reduction
    int maxi = 0, count = 0;
    //variable for time stamp
    struct timeval tv1, tv2;
    //variables for checking the a number is prime or not
    
    //end of declaration
    //starting timer here
    gettimeofday(&tv1, NULL);
	//MPI declaration ends here
	//Cuda declaration starts here
    int threadSize=10;
	if(number_of_prime<(2000*Num_Processor))
		threadSize=number_of_prime/(2*Num_Processor);
	else
		threadSize=1000;
	int	blocksize = (number_of_prime / (2*threadSize*Num_Processor));
	int totalthreads=blocksize*threadSize*Num_Processor;
	int arraycount[totalthreads];
	int *dev_count, *dev_max;
			for(int i=0;i<totalthreads;i++)
				arraycount[i]=0;
	hipMalloc((void **) &dev_count, totalthreads* sizeof(int));
    hipMalloc((void **) &dev_max, sizeof (int));
	//end of cuda declaration

    if (number_of_prime > 10) {
	    hipMemcpy(dev_max, &maxi, sizeof (int), hipMemcpyHostToDevice);
		hipMemcpy(dev_count, arraycount, totalthreads*sizeof (int), hipMemcpyHostToDevice);
        printf("number of blocks and threads %d / %d\n", blocksize, threadSize);
        cudeprime <<<blocksize, threadSize >>>(number_of_prime,dev_count, dev_max,p_id,Num_Processor);
        hipMemcpy(&maxi, dev_max, sizeof (int), hipMemcpyDeviceToHost);
		hipMemcpy( arraycount,dev_count, totalthreads*sizeof (int), hipMemcpyDeviceToHost);
		gettimeofday(&tv2, NULL);
		for(int i=0;i<totalthreads;i++)
		{
			if(arraycount[i]>0)
			{
				count++;
				if(maxi<arraycount[i])maxi=arraycount[i];
				//printf("%d is n and index %d\n",arraycount[i],i);
			}
		}
		
    } else {
        count = 4;
        maxi = 7;
    }
	
    //to broadcast he problem to other nodes from the root node

	
    
	
    printf("Local count = %d \t Local max value = %d\ncuda exit\n", count, maxi);
	
    hipFree(dev_max);
    hipFree(dev_count);

    
	returnval[0]=count;
	returnval[1]=maxi;
    return returnval;
}
